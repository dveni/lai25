
#include <hip/hip_runtime.h>
#include <stdio.h>


# define EL_PER_THREAD 4
# define BLOCKSIZE 256

__global__ void square_weird1(const float* in_array, float* out_array, const int N) {
    const int threadid = threadIdx.x;
    const int blockid = blockIdx.x;
    const int blocksize = blockDim.x;
    const int globalid = (blockid * blocksize + threadid) * EL_PER_THREAD;

    if (globalid >= N) {
        return; // Out of bounds
    }

    float in[EL_PER_THREAD];

    for (int i = 0; i < EL_PER_THREAD; i++) { //assume N is a multiple of 4
        in[i] = in_array[globalid + i];
    }

    for (int i = 0; i < EL_PER_THREAD; i++) { //assume N is a multiple of 4
        in[i] = in[i]*in[i];
    }

    for (int i = 0; i < EL_PER_THREAD; i++) { //assume N is a multiple of 4
        out_array[globalid + i] = in[i];
    }
}


int main() {
    const int N = 1024*1024*1024;
    float *in_array, *out_array;
    float *d_in_array, *d_out_array;

    // Allocate host memory
    in_array = (float*) malloc(N * sizeof(float));
    out_array = (float*) malloc(N * sizeof(float));

    // Initialize input array
    for (int i = 0; i < N; i++) {
        in_array[i] = static_cast<float>(i);
    }

    // Allocate device memory
    hipMalloc((void**)&d_in_array, N * sizeof(float));
    hipMalloc((void**)&d_out_array, N * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_in_array, in_array, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block and enough blocks to cover the array
    int num_blocks = (N + (BLOCKSIZE*EL_PER_THREAD)-1) / (BLOCKSIZE*EL_PER_THREAD);
    square_weird1<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);
    square_weird1<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);
    square_weird1<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    square_weird1<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }


    // Free device memory
    hipFree(d_in_array);
    hipFree(d_out_array);

    // Free host memory
    free(in_array);
    free(out_array);

    return 0;
}