
#include <hip/hip_runtime.h>
#include <stdio.h>


# define EL_PER_THREAD 4
# define BLOCKSIZE 256


__global__ void square_uncoalesced(const float* in_array, float* out_array, const unsigned int N) {
    const int threadid = threadIdx.x;
    const int blockid = blockIdx.x;
    const int blocksize = blockDim.x;
    const int globalid = (blockid * blocksize + threadid) * EL_PER_THREAD;

    if (globalid >= N) {
        return; // Out of bounds
    }

    // assume N is a multiple of 4
    float4 in;
    in = *reinterpret_cast<const float4*>(&in_array[globalid]);

    in.x = in.x * in.x;
    in.y = in.y * in.y;
    in.z = in.z * in.z;
    in.w = in.w * in.w;

    *reinterpret_cast<float4*>(&out_array[globalid]) = in;
}


int main() {
    const unsigned int N = 1024*1024*1024;
    float *in_array, *out_array;
    float *d_in_array, *d_out_array;

    // Allocate host memory
    in_array = (float*) malloc(N * sizeof(float));
    out_array = (float*) malloc(N * sizeof(float));

    // Initialize input array
    for (int i = 0; i < N; i++) {
        in_array[i] = static_cast<float>(i);
    }

    // Allocate device memory
    hipMalloc((void**)&d_in_array, N * sizeof(float));
    hipMalloc((void**)&d_out_array, N * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_in_array, in_array, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block and enough blocks to cover the array
    unsigned int num_blocks = (N + (BLOCKSIZE*EL_PER_THREAD)-1) / (BLOCKSIZE*EL_PER_THREAD);
    square_uncoalesced<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);
    square_uncoalesced<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);
    square_uncoalesced<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    square_uncoalesced<<<num_blocks, BLOCKSIZE>>>(d_in_array, d_out_array, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }


    // Free device memory
    hipFree(d_in_array);
    hipFree(d_out_array);

    // Free host memory
    free(in_array);
    free(out_array);

    return 0;
}