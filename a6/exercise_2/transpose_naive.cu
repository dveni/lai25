
#include <hip/hip_runtime.h>
#include <stdio.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 1;

__global__ void transposeNaive(const float *idata, float *odata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS){
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
  }
}

int main(){
    const int N = 1024*32;
    const int M = 1024*32;

    float *in_array, *out_array;
    float *d_in_array, *d_out_array;

    // Allocate host memory
    in_array = (float*) malloc(N * M * sizeof(float));
    out_array = (float*) malloc(N * M * sizeof(float));

    // Initialize input array
    for (int i = 0; i < N * M; i++) {
        in_array[i] = static_cast<float>(i);
    }

    // Allocate device memory
    hipMalloc((void**)&d_in_array, N * M * sizeof(float));
    hipMalloc((void**)&d_out_array, N * M * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_in_array, in_array, N * M * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block and enough blocks to cover the array
    dim3 dimGrid(N/TILE_DIM, M/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
    transposeNaive<<<dimGrid, dimBlock>>>(d_in_array, d_out_array);
    transposeNaive<<<dimGrid, dimBlock>>>(d_in_array, d_out_array);
    transposeNaive<<<dimGrid, dimBlock>>>(d_in_array, d_out_array);

    // make sure the kernels have finished executing before timing
    hipDeviceSynchronize();

    // time the kernel execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    transposeNaive<<<dimGrid, dimBlock>>>(d_in_array, d_out_array);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken for kernel execution: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    // Check for errors in kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error launching kernel: %s\n", hipGetErrorString(err));
        return -1;
    }
    // Synchronize device
    hipDeviceSynchronize();

    // Copy output data back to host
    hipMemcpy(out_array, d_out_array, N * M * sizeof(float), hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < N; i++) {
      for (int j=0; j < M; j++){
          if (out_array[i*M + j] != in_array[j*N + i]) {
              printf("Mismatch at index %d: %f != %f\n", i, out_array[i*M + j], in_array[j*N + i]);
              break;
          }
      }
    }

    // Free device memory
    hipFree(d_in_array);
    hipFree(d_out_array);

    // Free host memory
    free(in_array);
    free(out_array);

    return 0;
}